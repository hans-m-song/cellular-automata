#include "hip/hip_runtime.h"
#include "grid.hpp"

#include <stdlib.h>
#include <unistd.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "metric.hpp"
#include "util.hpp"

Metric metric;

int* generation;
int* next_generation;

void checkError(hipError_t e) {
  if (e != hipSuccess) {
    std::cerr << "CUDA error: " << int(e) << ": " 
              << hipGetErrorString(e) << "\n";
    abort();
  }
}

int empty_cell(int* generation, int width, int height) {
  int x, y, point;
  for (int i = 0; i < width * height; i++) {
    x = random(0, width - 1);
    y = random(0, height - 1);
    point = x * width + y;
    if (!generation[point]) {
      return point;
    }
  }

  return 0;
}

void init(int width, int height, double initial_density) {
  int size = width * height;

  int* init_generation = new int[size];
  
  for (int i = 0; i < size; i++) {
    init_generation[i] = 0;
  }
  
  int initial_cells = (int)(initial_density * size);
  
  int point;
  for (int i = 0; i < initial_cells; i++) {
    point = empty_cell(init_generation, width, height);
    init_generation[point] = 1;
  }

  // allocate memory on device
  checkError(hipMalloc((void**)&generation, size));
  checkError(hipMalloc((void**)&next_generation, size));

  // copy over empty data
  checkError(hipMemcpy(generation, init_generation, size, hipMemcpyHostToDevice));
  // don't need to initialize next_generation since it'll be overwritten during tick
  delete[] init_generation;
}

void deinit() {
  hipFree(generation);
  hipFree(next_generation);
}

__device__
int apply_direction(int width, int height, int i, int j, Direction direction) {
  int x = i;
  int y = j;
  switch (direction) {
    case Direction::N:
      x += 1;
      break;
    case Direction::NE:
      x += 1;
      y += 1;
      break;
    case Direction::E:
      y += 1;
      break;
    case Direction::SE:
      x -= 1;
      y += 1;
      break;
    case Direction::S:
      x -= 1;
      break;
    case Direction::SW:
      x -= 1;
      y -= 1;
      break;
    case Direction::W:
      y -= 1;
      break;
    case Direction::NW:
      x += 1;
      y -= 1;
      break;
  }

  return ((x + width) % width) * width + ((y + height) % height);
}

__device__
int sum_neighbour(int* generation, int width, int height, int x, int y) {
  int sum = 0;
  int adjacent;
  // check each direction
  for (int i = Direction::N; i <= Direction::NW; i++) {
    adjacent = apply_direction(width, height, x, y, (Direction)i);
    if (generation[adjacent]) {
      sum += 1;
    }
  }
  return sum;
}

__device__
void tick(int* generation, int* next_generation, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x_stride = blockDim.x * gridDim.x;
  int y_stride = blockDim.y * gridDim.y;

  for (int i = x; i < width; i += x_stride) {
    for (int j = y; j < height; j += y_stride) {
      int neighbours = sum_neighbour(generation, width, height, i, j);
      if (neighbours == 3) {
        // if 3 live neighbours exactly, cell lives
        next_generation[i * width + j] = 1;
      } else if (neighbours == 2) {
        // if 2 live neighbours exactly, cell maintains status
        next_generation[i * width + j] = generation[i * width + j];
      } else {
        // otherwise else cell dies
        next_generation[i * width + j] = 0;
      }
    }
  }
  // swap currently active generation
  auto temp = generation;
  generation = next_generation;
  next_generation = temp;
}

__device__
void print(int* generation, int width, int height) {
#ifdef VISUAL
#ifndef DEBUG
  printf("\e[1;1H\e[2J");
#endif
#endif
  
  for (int x = 0; x < width; x++) {
    for (int y = 0; y < height; y++) {
      printf(generation[x * width + y] ? "o " : ". ");
    }
    printf("\n");
  }
}

__global__
void run(int* generation, int* next_generation, int width, int height, int ticks) {
  for (int i = 0; i < ticks; i++) {
    __syncthreads(); // ensure generation is completely generated
    tick(generation, next_generation, width, height);
#ifdef VISUAL
    print(generation, width, height);
#endif
  }
}

Metric run_grid(int width, int height, double initial_density, int ticks) {
  metric.start(Measure::Total);
  
  metric.start(Measure::Init);
  init(width, height, initial_density);
  metric.stop(Measure::Init);

  metric.start(Measure::Run);
  run<<<1, 1>>>(generation, next_generation, width, height, ticks);
  metric.stop(Measure::Run);
  
  metric.stop(Measure::Total);

  deinit();

  return metric;
}
