#include "hip/hip_runtime.h"
#include "grid.hpp"

#include <stdlib.h>
#include <unistd.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "metric.hpp"
#include "util.hpp"

int width;
int height;
int size;
int* generation;
int* d_generation
int* next_generation;
int* d_next_generation
Metric metric;

void checkError(hipError_t e) {
   if (e != hipSuccess) {
      std::cerr << "CUDA error: " << int(e) << ": " 
                << hipGetErrorString(e) << "\n";
      abort();
   }
}

void init(double initial_density) {
  metric.start(Measure::Total);
  metric.start(Measure::Init);
  
  generation = allocate_space();
  next_generation = allocate_space();
  
  int initial_cells = (int)(initial_density * size);
  
  Point point;
  for (int i = 0; i < initial_cells; i++) {
    point = empty_cell();
    generation[point.first * width + point.second] = 1;
  }
  
  // allocate memory on device
  checkError(hipMalloc((void**)&d_generation, size));
  checkError(hipMalloc((void**)&d_next_generation, size));

  // copy over empty data
  checkError(hipMemcpy(d_generation, generation, size, hipMemcpyHostToDevice));
  checkError(hipMemcpy(d_next_generation, next_generation, size, hipMemcpyHostToDevice));

  metric.stop(Measure::Init);
}

void deinit() {
  delete[] generation;
  delete[] next_generation;
  hipFree(d_generation);
  hipFree(d_next_generation);
}

int* allocate_space(void) {
  int* space = new int[size];
  for (int x = 0; x < width; x++) {
    for (int y = 0; y < height; y++) {
      space[x * width + y] = 0;
    }
  }
  return space;
}


Point empty_cell(void) {
  int x, y;
  for (int i = 0; i < size; i++) {
    x = random(0, width - 1);
    y = random(0, height - 1);
    if (!generation[x * width + y]) {
      return Point(x, y);
    }
  }

  return Point(-1, -1);
}

__device__
int sum_neighbour(int x, int y) {
  int sum = 0;
  Point origin = Point(x, y);
  Point adjacent;
  // check each direction
  for (int i = Direction::N; i <= Direction::NW; i++) {
    adjacent = apply_direction(origin, (Direction)i);
    if (d_generation[adjacent.first * width + adjacent.second]) {
      sum += 1;
    }
  }
  return sum;
}

void print(void) {
#ifndef DEBUG
  std::cout << "\e[1;1H\e[2J";
#endif

  for (int x = 0; x < width; x++) {
    for (int y = 0; y < height; y++) {
      std::cout << (d_generation[x * width + y] ? "o " : ". ");
    }
    std::cout << "\n";
  }
}

__global__
void run(int ticks) {
  for (int i = 0; i < ticks; i++) {
    __syncthreads(); // ensure generation is completely generated
    tick();
  }
}

__global__
void tick(void) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x_stride = blockDim.x * gridDim.x;
  int y_stride = blockDim.y * gridDim.y;

  for (int i = x; i < N; i += x_stride) {
    for (int j = y; j < N; j += y_stride) {
      int neighbours = sum_neighbour(i, j);
      if (neighbours == 3) {
        // if 3 live neighbours exactly, cell lives
        d_next_generation[i * width + j] = 1;
      } else if (neighbours == 2) {
        // if 2 live neighbours exactly, cell maintains status
        d_next_generation[i * width + j] = d_generation[i * width + j];
      } else {
        // otherwise else cell dies
        d_next_generation[i * width + j] = 0;
      }
    }
  }
  // swap currently active generation
  std::swap(generation, next_generation);
}

__device__
Point apply_direction(Point origin, Direction direction) {
  int x = origin.first;
  int y = origin.second;
  switch (direction) {
    case Direction::N:
      x += 1;
      break;
    case Direction::NE:
      x += 1;
      y += 1;
      break;
    case Direction::E:
      y += 1;
      break;
    case Direction::SE:
      x -= 1;
      y += 1;
      break;
    case Direction::S:
      x -= 1;
      break;
    case Direction::SW:
      x -= 1;
      y -= 1;
      break;
    case Direction::W:
      y -= 1;
      break;
    case Direction::NW:
      x += 1;
      y -= 1;
      break;
  }

  return Point((x + width) % width, (y + height) % height);
}

Metric run_grid(int h_width, int h_height, double initial_density, int ticks) {
  width = h_width;
  height = h_height;
  size = width * height;

  init(initial_density);

  metric.start(Measure::Run);
  run<<<1, 1>>>(ticks);
  metric.stop(Measure::Run);
  metric.stop(Measure::Total);
  
  deinit();

  return metric;
}
