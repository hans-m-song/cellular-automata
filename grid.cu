#include "hip/hip_runtime.h"
#include "grid.hpp"

#include <stdlib.h>
#include <unistd.h>

#include <algorithm>
#include <iostream>
#include <vector>

#include "metric.hpp"
#include "util.hpp"

Metric metric;
int* generation;
int* next_generation;

int* d_generation;
int* d_next_generation;

int h_width;
int h_height;

__device__ int width;
__device__ int height;

void checkError(hipError_t e) {
  if (e != hipSuccess) {
    std::cerr << "CUDA error: " << int(e) << ": " 
              << hipGetErrorString(e) << "\n";
    abort();
  }
}

int empty_cell(int* generation, int width, int height) {
  int x, y, point;
  for (int i = 0; i < width * height; i++) {
    x = random(0, width - 1);
    y = random(0, height - 1);
    point = x * width + y;
    if (!generation[point]) {
      return point;
    }
  }

  return 0;
}

void init(int width, int height, double initial_density) {
  checkError(hipMemcpyToSymbol(HIP_SYMBOL("width"), &width, sizeof(int)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL("height"), &height, sizeof(int)));

  int size = width * height;

  generation = new int[size];
  next_generation = new int[size];
  
  for (int i = 0; i < size; i++) {
    generation[i] = 0;
  }
  
  int initial_cells = (int)(initial_density * size);
  
  int point;
  for (int i = 0; i < initial_cells; i++) {
    point = empty_cell(generation, width, height);
    generation[point] = 1;
  }
  
  // allocate memory on device
  checkError(hipMalloc((void**)&d_generation, size));
  checkError(hipMalloc((void**)&d_next_generation, size));

  // copy over empty data
  checkError(hipMemcpy(d_generation, generation, size, hipMemcpyHostToDevice));
  // don't need to initialize next_generation since it'll be overwritten during tick
}

void deinit() {
  delete[] generation;
  delete[] next_generation;
  hipFree(d_generation);
  hipFree(d_next_generation);
}

__device__
int apply_direction(int i, int j, Direction direction) {
  int x = i;
  int y = j;
  switch (direction) {
    case Direction::N:
      x += 1;
      break;
    case Direction::NE:
      x += 1;
      y += 1;
      break;
    case Direction::E:
      y += 1;
      break;
    case Direction::SE:
      x -= 1;
      y += 1;
      break;
    case Direction::S:
      x -= 1;
      break;
    case Direction::SW:
      x -= 1;
      y -= 1;
      break;
    case Direction::W:
      y -= 1;
      break;
    case Direction::NW:
      x += 1;
      y -= 1;
      break;
  }

  return ((x + width) % width) * width + ((y + height) % height);
}

__device__
int sum_neighbour(int* d_generation, int x, int y) {
  int sum = 0;
  int adjacent;
  // check each direction
  for (int i = Direction::N; i <= Direction::NW; i++) {
    adjacent = apply_direction(x, y, (Direction)i);
    if (d_generation[adjacent]) {
      sum += 1;
    }
  }
  return sum;
}

__device__
void tick(int* d_generation, int* d_next_generation) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x_stride = blockDim.x * gridDim.x;
  int y_stride = blockDim.y * gridDim.y;

  for (int i = x; i < N; i += x_stride) {
    for (int j = y; j < N; j += y_stride) {
      int neighbours = sum_neighbour(d_generation, i, j);
      if (neighbours == 3) {
        // if 3 live neighbours exactly, cell lives
        d_next_generation[i * width + j] = 1;
      } else if (neighbours == 2) {
        // if 2 live neighbours exactly, cell maintains status
        d_next_generation[i * width + j] = d_generation[i * width + j];
      } else {
        // otherwise else cell dies
        d_next_generation[i * width + j] = 0;
      }
    }
  }
  // swap currently active generation
  auto temp = d_generation;
  d_generation = d_next_generation;
  d_next_generation = temp;
}

__global__
void run(int ticks, int* d_generation, int* d_next_generation) {
  for (int i = 0; i < ticks; i++) {
    __syncthreads(); // ensure generation is completely generated
    tick(d_generation, d_next_generation);
  }
}

void print(void) {
#ifndef DEBUG
  std::cout << "\e[1;1H\e[2J";
#endif

  for (int x = 0; x < h_width; x++) {
    for (int y = 0; y < h_height; y++) {
      std::cout << (d_generation[x * h_width + y] ? "o " : ". ");
    }
    std::cout << "\n";
  }
}

Metric run_grid(int width, int height, double initial_density, int ticks) {
  h_width = width;
  h_height = height;
  metric.start(Measure::Total);

  metric.start(Measure::Init);
  init(width, height, initial_density);
  metric.stop(Measure::Init);

  metric.start(Measure::Run);
  run<<<1, 1>>>(ticks, d_generation, d_next_generation);
  metric.stop(Measure::Run);
  
  metric.stop(Measure::Total);

  deinit();

  return metric;
}
